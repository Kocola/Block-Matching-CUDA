#include "hip/hip_runtime.h"
#include "DivideBlock3D.h"
#include "Parameter.h"
#include "ReadSaveImage.h"

#include <hip/hip_runtime.h>
#include <>

/*
* ���ܣ���ͼ������ȡ��
* ���룺im_D ����ȡ���ͼ��
* �����blocks_D ��ȡ�Ŀ飬һ��Ϊһ����
* ���룺row ����ͼ������
* ���룺col ����ͼ������
*/
__global__ void DivideBlock3D_raw(float *im_D, float *blocks_D, int row, int col)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������
	int index = y_id * col + x_id;

	__shared__ float sData[BLOCKSIZE + 4][BLOCKSIZE + 4];

	for (int k = 0; k < batch; k++)
	{
		float *dataPtr = &im_D[k * row * col];

		/* ���Ͻ� 16 * 16 */
		if (x_id < col && y_id < row)
			sData[threadIdx.y][threadIdx.x] = dataPtr[index];

		if (blockDim.x != gridDim.x && blockDim.y != gridDim.y)
		{
			/* ���½� 4 * 4 */
			if (threadIdx.y >= 12 && threadIdx.x >= 12)
				sData[threadIdx.y + 4][threadIdx.x + 4] = dataPtr[index + 4 * col + 4];

			/* ���Ͻ� 16 * 4 */
			if (threadIdx.x >= 12)
				sData[threadIdx.y][threadIdx.x + 4] = dataPtr[index + 4];

			/* ���½� 4 * 16 */
			if (threadIdx.y >= 12)
				sData[threadIdx.y + 4][threadIdx.x] = dataPtr[index + 4 * col];
		}

		__syncthreads();

		if (x_id < col - 4 && y_id < row - 4)
		{
			int indexOffset = (y_id * (col - blockR + 1) + x_id) * blockSizes + k * blockR * blockR;

			for (int i = 0; i < blockR; i++)
			{
				int indexRow = i * blockR;
				for (int j = 0; j < blockR; j++)
				{
					blocks_D[indexOffset + indexRow + j] = sData[threadIdx.y + i][threadIdx.x + j];
				}
			}
		}

		__syncthreads();
	}
}


/*
* ���ܣ���ͼ������ȡ��
* ���룺im_D ����ȡ���ͼ��
* �����blocks_D ��ȡ�Ŀ飬һ��Ϊһ����
* ���룺row ����ͼ������
* ���룺col ����ͼ������
*/
__global__ void DivideBlock3D(float *im_D, float *blocks_D, int row, int col)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������
	int index = y_id * col + x_id;

	__shared__ float sData[BLOCKSIZE + (blockR - 1)][BLOCKSIZE + (blockR - 1)];


	for (int k = 0; k < batch; k++)
	{
		float *dataPtr = &im_D[k * row * col];

		/* ���Ͻ� 16 * 16 */
		if (x_id < col && y_id < row) 
			sData[threadIdx.y][threadIdx.x] = dataPtr[index];

		if (blockDim.x != gridDim.x && blockDim.y != gridDim.y)
		{
			/* ���½� 4 * 4 */
			if (threadIdx.y >= BLOCKSIZE - (blockR - 1) && threadIdx.x >= BLOCKSIZE - (blockR - 1))
				sData[threadIdx.y + (blockR - 1)][threadIdx.x + (blockR - 1)] = dataPtr[index + (blockR - 1) * col + (blockR - 1)];

			/* ���Ͻ� 16 * 4 */
			if (threadIdx.x >= BLOCKSIZE - (blockR - 1))
				sData[threadIdx.y][threadIdx.x + (blockR - 1)] = dataPtr[index + (blockR - 1)];

			/* ���½� 4 * 16 */
			if (threadIdx.y >= BLOCKSIZE - (blockR - 1))
				sData[threadIdx.y + (blockR - 1)][threadIdx.x] = dataPtr[index + (blockR - 1) * col];
		}

		__syncthreads();

		if (x_id < col - blockR + 1 && y_id < row - blockR + 1)
		{
#if majorRow
			int indexOffset = (y_id * (col - blockR + 1) + x_id) * blockSizes + k * blockR * blockR; // ȡ�鰴�����ȣ�(0,1)��ʼ���ǵ�2��
#else
			int indexOffset = (x_id * (row - blockR + 1) + y_id) * blockSizes + k * blockR * blockR; // ȡ�鰴�����ȣ�(1,0)��ʼ���ǵ�2��
#endif

			for (int i = 0; i < blockR; i++)
			{
				int indexRow = i * blockR;
				for (int j = 0; j < blockR; j++)
				{
					blocks_D[indexOffset + indexRow + j] = sData[threadIdx.y + i][threadIdx.x + j];
				}
			}
		}

		__syncthreads();
	}
}

