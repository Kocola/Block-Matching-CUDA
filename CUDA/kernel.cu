#include "Parameter.h"
#include "ReadSaveImage.h"

#include <hip/hip_runtime.h>
#include <>

#include <thrust\device_vector.h>
#include <thrust\host_vector.h>
#include <thrust\sequence.h>

#include "SetSearchWinCoordinate.h"
#include "DivideBlock3D.h"
#include "BlockMatching.h"

#include <iostream>
#include <vector>

using std::cout;
using std::endl;
using std::string;
using std::vector;


void CUDAmain(float *im_H)
{
	/* ��ʼ������ */
	vector<int> leftUpRow; // ÿ�����Ŀ������ʼ����
	vector<int> leftUpCol; // ÿ�����Ŀ������ʼ����

	SetParameter(imRow, imCol, winStep, blockR, leftUpRow, leftUpCol);

	int N1 = leftUpRow.size(); // ���������Ŀ�
	int M1 = leftUpCol.size(); // ���������Ŀ�

	int N = imRow - blockR + 1; // ������ȫ�ֿ�
	int M = imCol - blockR + 1; // ������ȫ�ֿ�


	/* �����������ڴ� */
	float *blocks_H = (float*)malloc(allBlockNum * blockSizes * sizeof(float));

	/* �����豸���ڴ� */
	float *im_D;
	hipMalloc((void**)&im_D, imRow * imCol * batch * sizeof(float));

	float *blocks_D;
	hipMalloc((void**)&blocks_D, allBlockNum * blockSizes * sizeof(float));
	hipMemset(blocks_D, 0, allBlockNum * blockSizes * sizeof(float));
	
	int *leftUpRow_D, *leftUpCol_D;
	hipMalloc((void**)&leftUpRow_D, N1 * sizeof(int));
	hipMalloc((void**)&leftUpCol_D, M1 * sizeof(int));

	int *rmin_D, *rmax_D, *cmin_D, *cmax_D;
	hipMalloc((void**)&rmin_D, N1 * sizeof(int));
	hipMalloc((void**)&rmax_D, N1 * sizeof(int));
	hipMalloc((void**)&cmin_D, M1 * sizeof(int));
	hipMalloc((void**)&cmax_D, M1 * sizeof(int));

	int *I_D;
	hipMalloc((void**)&I_D, N * M * sizeof(int));

	int *posIdx_D;
	hipMalloc((void**)&posIdx_D, N1 * M1 * similarBlkNum * sizeof(int));
	hipMemset(posIdx_D, 0, N1 * M1 * similarBlkNum * sizeof(int));

	float *weiIdx_D;
	hipMalloc((void**)&weiIdx_D, N1 * M1 * similarBlkNum * sizeof(float));
	hipMemset(weiIdx_D, 0, N1 * M1 * similarBlkNum * sizeof(float));


	/* �������� */
	hipMemcpy(im_D, im_H, imRow * imCol * batch * sizeof(float), hipMemcpyHostToDevice);

	hipMemcpy(leftUpRow_D, &leftUpRow[0], N1 *sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(leftUpCol_D, &leftUpCol[0], M1 *sizeof(int), hipMemcpyHostToDevice);

	/* �����̸߳���߳̿� */
	dim3 dimBlock1D(BLOCKSIZE * BLOCKSIZE);
	dim3 dimBlock2D(BLOCKSIZE, BLOCKSIZE);
	dim3 dimGrid1D_4(4);
	dim3 dimGrid2D_N_M((M + dimBlock2D.x - 1) / dimBlock2D.x, (N + dimBlock2D.y - 1) / dimBlock2D.y);
	dim3 dimGrid2D_N1_M1((M1 + dimBlock2D.x - 1) / dimBlock2D.x, (N1 + dimBlock2D.y - 1) / dimBlock2D.y);
	dim3 dimGrid1D_N1M1((N1 * M1 + dimBlock1D.x - 1) / dimBlock1D.x);


	/* ��¼ʱ�� */
	hipEvent_t start_GPU, end_GPU;
	float elaspsedTime;
	hipEventCreate(&start_GPU);
	hipEventCreate(&end_GPU);
	hipEventRecord(start_GPU, 0);

	SetSearchWinCoordinate<<<dimGrid1D_4, dimBlock1D>>>(leftUpRow_D, leftUpCol_D, rmin_D, rmax_D, cmin_D, cmax_D, N1, M1, (N - 1), (M - 1));

	/* ����Ԫ��ָ�뵽 device_vector ָ�� */
	thrust::device_ptr<int> I_D_ptr(I_D);

	/* ���� 0 - (N * M) �����������ڰ���ֵ���򼰺ϲ� */
	thrust::sequence(I_D_ptr, I_D_ptr + (N * M));

	/* ��ʱ���� */
	hipEventRecord(end_GPU, 0);
	hipEventSynchronize(end_GPU);
	hipEventElapsedTime(&elaspsedTime, start_GPU, end_GPU);

	/* ��ӡ��Ϣ */
	std::cout << "��ʼ����ʱ��Ϊ��" << elaspsedTime << "ms." << std::endl;

	/*****************************************************************************************************************************************************/

	/* ��¼ʱ�� */
	elaspsedTime = 0.0f;
	hipEventCreate(&start_GPU);
	hipEventCreate(&end_GPU);
	hipEventRecord(start_GPU, 0);

	DivideBlock3D<<<dimGrid2D_N_M, dimBlock2D>>>(im_D, blocks_D, imRow, imCol);

	/* ��ʱ���� */
	hipEventRecord(end_GPU, 0);
	hipEventSynchronize(end_GPU);
	hipEventElapsedTime(&elaspsedTime, start_GPU, end_GPU);

	/* ��ӡ��Ϣ */
	std::cout << "DivideBlock3D ��ʱ��Ϊ��" << elaspsedTime << "ms." << std::endl;

	/*****************************************************************************************************************************************************/
	/* ��¼ʱ�� */
	elaspsedTime = 0.0f;
	hipEventCreate(&start_GPU);
	hipEventCreate(&end_GPU);
	hipEventRecord(start_GPU, 0);

	//BlockMatching_R<<<dimGrid2D_N1_M1, dimBlock2D>>>(blocks_D, leftUpRow_D, leftUpCol_D, rmin_D, rmax_D, cmin_D, cmax_D, posIdx_D, weiIdx_D, N1, M1);
	BlockMatching_S<<<dimGrid2D_N1_M1, dimBlock2D>>>(blocks_D, leftUpRow_D, leftUpCol_D, posIdx_D, weiIdx_D, N1, M1);
	//BlockMatching_RS<<<dimGrid2D_N1_M1, dimBlock2D>>>(blocks_D, leftUpRow_D, leftUpCol_D, rmin_D, rmax_D, cmin_D, cmax_D, posIdx_D, weiIdx_D, N1, M1);

	/* ��ʱ���� */
	hipEventRecord(end_GPU, 0);
	hipEventSynchronize(end_GPU);
	hipEventElapsedTime(&elaspsedTime, start_GPU, end_GPU);

	/* ��ӡ��Ϣ */
	std::cout << "BlockMatching ��ʱ��Ϊ��" << elaspsedTime << "ms." << std::endl;

	/*****************************************************************************************************************************************************/
	/* ��¼ʱ�� */
	elaspsedTime = 0.0f;
	hipEventCreate(&start_GPU);
	hipEventCreate(&end_GPU);
	hipEventRecord(start_GPU, 0);

	BM_Calculate_Weight<<<dimGrid1D_N1M1, dimBlock1D>>>(weiIdx_D, N1, M1);

	/* ��ʱ���� */
	hipEventRecord(end_GPU, 0);
	hipEventSynchronize(end_GPU);
	hipEventElapsedTime(&elaspsedTime, start_GPU, end_GPU);

	/* ��ӡ��Ϣ */
	std::cout << "����Ȩ�ص�ʱ��Ϊ��" << elaspsedTime << "ms." << std::endl;

	/*****************************************************************************************************************************************************/


	/* �ͷ��豸���ڴ�*/
	hipFree(im_D);
	hipFree(blocks_D);
	hipFree(leftUpRow_D);
	hipFree(leftUpCol_D);
	hipFree(rmin_D);
	hipFree(rmax_D);
	hipFree(cmin_D);
	hipFree(cmax_D);
	hipFree(weiIdx_D);
	hipFree(posIdx_D);
	hipFree(I_D);
	

	/* �ͷ��������ڴ� */
	free(blocks_H);
}

int main()
{
	string strIm = "D:\\Document\\vidpic\\CUDA\\BlockMatch\\im.txt";
	float *im_H = (float*)malloc(imRow * imCol * batch * sizeof(float));
	ReadFile(im_H, strIm, imRow * imCol * batch);

	CUDAmain(im_H);

	free(im_H);

	hipDeviceReset();
	return 0;
}