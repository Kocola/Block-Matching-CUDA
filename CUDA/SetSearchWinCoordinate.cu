#include "hip/hip_runtime.h"
#include "SetSearchWinCoordinate.h"
#include "Parameter.h"

/*
* ���ܣ��������Ŀ���ʼ���귶Χ
* ���룺row ͼ������
* ���룺col ͼ������
* ���룺iBlock ��Ĵ�С
* ���룺leftUpRow �����ʼ������
* �����leftUpCol �����ʼ������
* ���룺im_H ͼ�񣬴洢˳����-��-ҳ
*/
void SetParameter(int row, int col, int step, int iBlock, std::vector<int> &leftUpRow, std::vector<int> &leftUpCol)
{
	int N = row - iBlock; // �����ʼ�����귶Χ
	int M = col - iBlock; // �����ʼ�����귶Χ

	for (int i = 0; i < N; i += step)
	{
		leftUpRow.push_back(i);
	}

	if (leftUpRow.back() != N)
	{
		leftUpRow.push_back(N);
	}

	for (int i = 0; i < M; i += step)
	{
		leftUpCol.push_back(i);
	}

	if (leftUpCol.back() != M)
	{
		leftUpCol.push_back(M);
	}
}

/*
* ���ܣ����������������귶Χ
* ���룺leftUpRow_D ÿ�����Ŀ������ʼ����
* ���룺leftUpCol_D ÿ�����Ŀ������ʼ����
* �����rmin_D ������������С����
* �����rmax_D �����������������
* �����cmin_D ������������С����
* �����cmax_D �����������������
* ���룺rowNum ���Ŀ������
* ���룺colNum ���Ŀ������
* ���룺rowMax ������������������꣬�� 0 ��ʼ
* ���룺colMax ������������������꣬�� 0 ��ʼ
*/
__global__ void SetSearchWinCoordinate(int *leftUpRow_D, int *leftUpCol_D, int *rmin_D, int *rmax_D, int *cmin_D, int *cmax_D, int rowNum, int colNum, int rowMax, int colMax)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x;

	if (0 == blockIdx.x) /* ������������С���� */
	{
		for (int i = threadIdx.x; i < rowNum; i += blockDim.x)
		{
			int currData = leftUpRow_D[i] - winRadius;
			rmin_D[i] = currData >= 0 ? currData : 0;
			//printf("rmin_D[%d] = %d\n", i, rmin_D[i]);
		}
	}
	else if (1 == blockIdx.x) /* ����������������� */
	{
		for (int i = threadIdx.x; i < rowNum; i += blockDim.x)
		{
			int currData = leftUpRow_D[i] + winRadius;
			rmax_D[i] = currData < rowMax ? currData : rowMax;
			//printf("rmax_D[%d] = %d\n", i, rmax_D[i]);
		}
	}
	else if (2 == blockIdx.x) /* ������������С���� */
	{
		for (int i = threadIdx.x; i < colNum; i += blockDim.x)
		{
			int currData = leftUpCol_D[i] - winRadius;
			cmin_D[i] = currData >= 0 ? currData : 0;
			//printf("cmin_D[%d] = %d\n", i, cmin_D[i]);
		}
	}
	else if (3 == blockIdx.x) /* ����������������� */
	{
		for (int i = threadIdx.x; i < colNum; i += blockDim.x)
		{
			int currData = leftUpCol_D[i] + winRadius;
			cmax_D[i] = currData < colMax ? currData : colMax;
			//printf("cmax_D[%d] = %d\n", i, cmax_D[i]);
		}
	}

}