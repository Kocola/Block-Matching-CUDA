#include "hip/hip_runtime.h"
#include "BlockMatching.h"
#include "Parameter.h"
#include <iostream>

__constant__ int searchOffset[2][8] = { { -1, -1, -1, 0, 0, 1, 1, 1 }, { -1, 0, 1, -1, 1, -1, 0, 1 } }; // ��������ƫ��

/**
* ���ܣ���������������ŷ������
* ���룺objects ��������
* �����clusters ������������
* ���룺vecLength ��������
*/
__device__ inline static float EuclidDistance(float *center, float *search, int vecLength)
{
	float dist = 0.0f;

	for (int i = 0; i < vecLength; i++)
	{
		float onePoint = center[i] - search[i];
		dist += onePoint * onePoint;
	}
	
	return(dist);
}

/**
* ���ܣ����ҵ�ǰ���Ƿ�Ϊ���ƿ�
* ���룺posIdx_D ��ǰ���Ŀ��Ӧ�����ƿ��λ������
* �����weiIdx_D ��ǰ���Ŀ��Ӧ�����ƿ��Ȩ������
* ���룺wei �µ�Ȩ��
* ���룺pos �µ�Ȩ�ض�Ӧ��λ��
*/
__device__ void FindSimilarBlocks(int *posIdx_D, float *weiIdx_D, float wei, int pos)
{
	int index = similarBlkNum - 1;

	while (index >= 0 && abs(weiIdx_D[index]) < 1e-6)
	{
		index--;
	}

	if (index == similarBlkNum - 1 && weiIdx_D[index] >= wei)
	{
		index--;
	}

	while (index >= 0 && weiIdx_D[index] > wei)
	{
		weiIdx_D[index + 1] = weiIdx_D[index];
		posIdx_D[index + 1] = posIdx_D[index];

		index--;
	}

	if (similarBlkNum - 1 != index)
	{
		weiIdx_D[index + 1] = wei;
		posIdx_D[index + 1] = pos;
	}
}

/**
*�����posIdx_D ���ƿ��λ��
* �����weiIdx_D ���ƿ��Ȩ��
* ���룺rowNum ���Ŀ������
* ���룺colNum ���Ŀ������
*/
__global__ void BM_Init_WeightAndPos(int *posIdx_D, float *weiIdx_D, int rowNum, int colNum)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������

	if (y_id < rowNum && x_id < colNum)
	{
		int offInCentralBlk = y_id * colNum + x_id; // ��������

		float *weiIdx = &weiIdx_D[offInCentralBlk * similarBlkNum];
		int *posIdx = &posIdx_D[offInCentralBlk * similarBlkNum];
		for (int i = 0; i < similarBlkNum; i++)
		{
			weiIdx[i] = 2e30;
			posIdx[i] = offInCentralBlk;
		}
	}
}

/*
* ���ܣ����������������귶Χ
* ���룺blocks_D ��ȡ�Ŀ飬һ��Ϊһ����
* ���룺leftUpRow_D ÿ�����Ŀ������ʼ����
* ���룺leftUpCol_D ÿ�����Ŀ������ʼ����
* ���룺rmin_D ������������С����
* ���룺rmax_D �����������������
* ���룺cmin_D ������������С����
* ���룺cmax_D �����������������
* ���룺blocksMean_D ÿ����ľ�ֵ
* ���룺blocksVar_D ÿ����ķ���
* �����posIdx_D ���ƿ��λ��
* �����weiIdx_D ���ƿ��Ȩ��
* ���룺rowNum ���Ŀ������
* ���룺colNum ���Ŀ������
*/
__global__ void BlockMatching_R(float *blocks_D, int *leftUpRow_D, int *leftUpCol_D, int *rmin_D, int *rmax_D, int *cmin_D, int *cmax_D, float *blocksMean_D, float *blocksVar_D, int *posIdx_D, float *weiIdx_D, int rowNum, int colNum)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������

	if (y_id < rowNum && x_id < colNum)
	{
		int offInAllBlk = leftUpRow_D[y_id] * (imCol - blockR + 1) + leftUpCol_D[x_id]; // ��������
		int offInCentralBlk = y_id * colNum + x_id; // ��������

		float *ptrCenter = &blocks_D[offInAllBlk * blockSizes];

		/* �������еĿ� */
		for (int i = rmin_D[y_id]; i <= rmax_D[y_id]; i++)
		{
			for (int j = cmin_D[x_id]; j <= cmax_D[x_id]; j++)
			{
				int searchIdx = i * (imCol - blockR + 1) + j; // ��������
				float *ptrSearchIdx = &blocks_D[searchIdx * blockSizes];

				if (BM_muMax > (blocksMean_D[offInAllBlk] / blocksMean_D[searchIdx]) > BM_muMin && BM_deltaMax > (blocksVar_D[offInAllBlk] / blocksVar_D[searchIdx]) > BM_deltaMin)
				{
					float dist = EuclidDistance(ptrCenter, ptrSearchIdx, blockSizes);
					FindSimilarBlocks(&posIdx_D[offInCentralBlk * similarBlkNum], &weiIdx_D[offInCentralBlk * similarBlkNum], dist / float(blockSizes), searchIdx);
				}
			}
		}
		//if (x_id == 0 && y_id == 252)
		//{
		//	for (int i = 0; i < similarBlkNum; i++)
		//	{
		//		printf("x_id = %d, y_id = %d, posIdx_D[%d] = %d, weiIdx_D[%d] = %f\n", x_id, y_id, i, posIdx_D[offInCentralBlk * similarBlkNum + i], i, weiIdx_D[offInCentralBlk * similarBlkNum + i]);
		//	}
		//}
	}
}

/*
* ���ܣ����������������귶Χ
* ���룺blocks_D ��ȡ�Ŀ飬һ��Ϊһ����
* ���룺leftUpRow_D ÿ�����Ŀ������ʼ����
* ���룺leftUpCol_D ÿ�����Ŀ������ʼ����
* ���룺blocksMean_D ÿ����ľ�ֵ
* ���룺blocksVar_D ÿ����ķ���
* �����posIdx_D ���ƿ��λ��
* �����weiIdx_D ���ƿ��Ȩ��
* ���룺rowNum ���Ŀ������
* ���룺colNum ���Ŀ������
*/
__global__ void BlockMatching_S(float *blocks_D, int *leftUpRow_D, int *leftUpCol_D, float *blocksMean_D, float *blocksVar_D, int *posIdx_D, float *weiIdx_D, int rowNum, int colNum)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������

	if (y_id < rowNum && x_id < colNum)
	{
		int offInAllBlk = leftUpRow_D[y_id] * (imCol - blockR + 1) + leftUpCol_D[x_id]; // ��������
		int offInCentralBlk = y_id * colNum + x_id; // ��������

		float *ptrCenter = &blocks_D[offInAllBlk * blockSizes];

		/* ����Jump Flooding �еĿ� */
		int searchIdx = offInAllBlk;
		for (int step = 1; step <= imRow / 2; step *= 2)
		{
			for (int i = 0; i < 8; i++)
			{
				int currRow = leftUpRow_D[y_id] + searchOffset[0][i] * step;
				int currCol = leftUpCol_D[x_id] + searchOffset[1][i] * step;
				if (currRow >= 0 && currRow < imRow - blockR && currCol >=0 && currCol < imCol - blockR)
				{
					searchIdx = currRow * (imCol - blockR + 1) + currCol; // ��������
					float *ptrSearchIdx = &blocks_D[searchIdx * blockSizes];

					if (BM_muMax >(blocksMean_D[offInAllBlk] / blocksMean_D[searchIdx]) > BM_muMin && BM_deltaMax >(blocksVar_D[offInAllBlk] / blocksVar_D[searchIdx]) > BM_deltaMin)
					{
						float dist = EuclidDistance(ptrCenter, ptrSearchIdx, blockSizes);
						FindSimilarBlocks(&posIdx_D[offInCentralBlk * similarBlkNum], &weiIdx_D[offInCentralBlk * similarBlkNum], dist / float(blockSizes), searchIdx);
					}
				}
			}
		}
		FindSimilarBlocks(&posIdx_D[offInCentralBlk * similarBlkNum], &weiIdx_D[offInCentralBlk * similarBlkNum], 0.0f, offInAllBlk);
	}
}

/*
* ���ܣ����������������귶Χ
* ���룺blocks_D ��ȡ�Ŀ飬һ��Ϊһ����
* ���룺leftUpRow_D ÿ�����Ŀ������ʼ����
* ���룺leftUpCol_D ÿ�����Ŀ������ʼ����
* ���룺rmin_D ������������С����
* ���룺rmax_D �����������������
* ���룺cmin_D ������������С����
* ���룺cmax_D �����������������
* ���룺blocksMean_D ÿ����ľ�ֵ
* ���룺blocksVar_D ÿ����ķ���
* �����posIdx_D ���ƿ��λ��
* �����weiIdx_D ���ƿ��Ȩ��
* ���룺rowNum ���Ŀ������
* ���룺colNum ���Ŀ������
*/
__global__ void BlockMatching_RS(float *blocks_D, int *leftUpRow_D, int *leftUpCol_D, int *rmin_D, int *rmax_D, int *cmin_D, int *cmax_D, float *blocksMean_D, float *blocksVar_D, int *posIdx_D, float *weiIdx_D, int rowNum, int colNum)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������

	if (y_id < rowNum && x_id < colNum)
	{
		int offInAllBlk = leftUpRow_D[y_id] * (imCol - blockR + 1) + leftUpCol_D[x_id]; // ��������
		int offInCentralBlk = y_id * colNum + x_id; // ��������

		float *ptrCenter = &blocks_D[offInAllBlk * blockSizes];

		/* �������еĿ� */
		int searchIdx = offInAllBlk;
		for (int i = rmin_D[y_id]; i <= rmax_D[y_id]; i++)
		{
			for (int j = cmin_D[x_id]; j <= cmax_D[x_id]; j++)
			{
				searchIdx = i * (imCol - blockR + 1) + j; // ��������
				float *ptrSearchIdx = &blocks_D[searchIdx * blockSizes];
				if (BM_muMax > (blocksMean_D[offInAllBlk] / blocksMean_D[searchIdx]) > BM_muMin && BM_deltaMax > (blocksVar_D[offInAllBlk] / blocksVar_D[searchIdx]) > BM_deltaMin)
				{
					float dist = EuclidDistance(ptrCenter, ptrSearchIdx, blockSizes);
					FindSimilarBlocks(&posIdx_D[offInCentralBlk * similarBlkNum], &weiIdx_D[offInCentralBlk * similarBlkNum], dist / float(blockSizes), searchIdx);
				}
			}
		}

		/* ����Jump Flooding �еĿ� */
		for (int step = winRadius + 1; step <= imRow / 2; step *= 2)
		{
			for (int i = 0; i < 8; i++)
			{
				int currRow = leftUpRow_D[y_id] + searchOffset[0][i] * step;
				int currCol = leftUpCol_D[x_id] + searchOffset[1][i] * step;
				if (currRow >= 0 && currRow < imRow - blockR && currCol >=0 && currCol < imCol - blockR)
				{
					searchIdx = currRow * (imCol - blockR + 1) + currCol; // ��������
					float *ptrSearchIdx = &blocks_D[searchIdx * blockSizes];

					if (BM_muMax >(blocksMean_D[offInAllBlk] / blocksMean_D[searchIdx]) > BM_muMin && BM_deltaMax >(blocksVar_D[offInAllBlk] / blocksVar_D[searchIdx]) > BM_deltaMin)
					{
						float dist = EuclidDistance(ptrCenter, ptrSearchIdx, blockSizes);
						FindSimilarBlocks(&posIdx_D[offInCentralBlk * similarBlkNum], &weiIdx_D[offInCentralBlk * similarBlkNum], dist / float(blockSizes), searchIdx);
					}
				}
			}
		}
	}
}

/*
* ���ܣ��������ƿ��Ȩ��
* �����weiIdx_D ���ƿ��Ȩ��
* ���룺rowNum ���Ŀ������
* ���룺colNum ���Ŀ������
*/
__global__ void BM_Calculate_Weight(float *weiIdx_D, int rowNum, int colNum)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������ 

	if (x_id < rowNum * colNum)
	{
		__shared__ float sData[BLOCKSIZE * BLOCKSIZE][similarBlkNum];

		float *currBlk = &weiIdx_D[x_id * similarBlkNum];
		float sum = 1e-15;
		
		if (x_id == 0 * colNum + 252)
		{
			for (int i = 0; i < similarBlkNum; i++)
			{
				printf("x_id = %d, sum = %f, weiIdx_D[%d] = %f\n", x_id, sum, i, weiIdx_D[x_id * similarBlkNum + i]);
			}
			printf("\n");
		}

		/* ��˹��Ȩ �� ���빲���ڴ�*/
		for (int i = 0; i < similarBlkNum; i++)
		{
			sData[threadIdx.x][i] = exp(-currBlk[i] / BM_hp);
		}

		__syncthreads();

		/* ��� */
		for (int i = 0; i < similarBlkNum; i++)
		{
			sum += sData[threadIdx.x][i];
		}
		__syncthreads();

		/* ���� */
		for (int i = 0; i < similarBlkNum; i++)
		{
			currBlk[i] = sData[threadIdx.x][i] / sum;
		}

		if (x_id == 0 * colNum + 252)
		{
			for (int i = 0; i < similarBlkNum; i++)
			{
				printf("x_id = %d, sum = %f, weiIdx_D[%d] = %f\n", x_id, sum, i, weiIdx_D[x_id * similarBlkNum + i]);
			}
		}
	}
}