#include "hip/hip_runtime.h"
#include "CalculateMeanVar.h"
#include "Parameter.h"

#include <hip/hip_runtime.h>
#include <>
/*
* ���ܣ�����ÿ��ͼ���ľ�ֵ�ͷ���
* ���룺blocks_D ��ȡ�Ŀ飬һ��Ϊһ����
* �����blocksMean_D ÿ����ľ�ֵ
* �����blocksVar_D ÿ����ķ���
* ���룺rowNum ȫ�ֿ������
* ���룺colNum ȫ�ֿ������
*/
__global__ void BM_Calculate_Mean_Var(float *blocks_D, float *blocksMean_D, float *blocksVar_D, int rowNum, int colNum)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������ 

	if (x_id < rowNum * colNum)
	{
		float *currBlk = &blocks_D[x_id * blockSizes];

		/* �����ֵ */
		float blocksMean = 0.0f;
		for (int i = 0; i < blockSizes; i++)
		{
			blocksMean += currBlk[i];
		}
		blocksMean = blocksMean / blockSizes;

		blocksMean_D[x_id] = blocksMean;

		/* ���㷽�� */
		float blocksVar = 0.0f;
		for (int i = 0; i < blockSizes; i++)
		{
			blocksVar += (currBlk[i] - blocksMean) * (currBlk[i] - blocksMean);
		}
		blocksVar = blocksVar / blockSizes;

		blocksVar_D[x_id] = blocksVar;
	}
}